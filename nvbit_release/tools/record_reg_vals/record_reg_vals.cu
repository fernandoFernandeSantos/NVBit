#include "hip/hip_runtime.h"
/* Copyright (c) 2019, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include <assert.h>
#include <stdint.h>
#include <stdio.h>
#include <unistd.h>
#include <string>
#include <map>
#include <vector>
#include <unordered_set>
#include <utility> // cbank list
#include <fstream> //final trace
#include <regex> // find cbank in the sass
#include <sstream>
/* every tool needs to include this once */
#include "nvbit_tool.h"

/* nvbit interface file */
#include "nvbit.h"

/* for channel */
#include "utils/channel.hpp"

/* contains definition of the reg_info_t structure */
#include "common.h"

/* Channel used to communicate from GPU to CPU receiving thread */
#define CHANNEL_SIZE (1l << 20)
static __managed__ ChannelDev channel_dev;
static ChannelHost channel_host;

/* receiving thread and its control variables */
pthread_t recv_thread;
volatile bool recv_thread_started = false;
volatile bool recv_thread_receiving = false;

/* skip flag used to avoid re-entry on the nvbit_callback when issuing
 * flush_channel kernel call */
bool skip_flag = false;

/* global control variables for this tool */
uint32_t instr_begin_interval = 0;
uint32_t instr_end_interval = UINT32_MAX;
int verbose = 0;

/* opcode to id map and reverse map  */
std::map<std::string, int> sass_to_id_map;
std::map<int, std::string> id_to_sass_map;

/**
 * Fernando mod
 * Final trace file
 */
constexpr char output_trace_file[] = "nvbit_trace_file.txt";
std::ofstream nvbit_trace_file;

/*get the c[bankid][bankoffset] list from the sass instruction*/
std::vector<std::pair<int32_t, int32_t>> extract_cbank_vector(const std::string& sass_line) {
	std::regex sass_regex(".*c\\[(0[xX][0-9a-fA-F]+)\\]\\[(0[xX][0-9a-fA-F]+)\\].*");
	std::smatch match;
	auto m = std::regex_match(sass_line, match, sass_regex);
	if (m == false && (sass_line.find("c[") != std::string::npos)) {
		std::cerr << "Problem when parsing the SASS line " << sass_line << std::endl;
		throw;
	}
	std::vector<std::pair<int32_t, int32_t>> cbank_list;
	for (uint32_t i = 1; i < match.size(); i += 2) {
		auto bank_id = std::stoi(match[i], nullptr, 16);
		auto bank_offset = std::stoi(match[i + 1], nullptr, 16);
		std::pair < int32_t, int32_t > cbank(bank_id, bank_offset);
		cbank_list.push_back(cbank);
	}
	return cbank_list;
}

/**************************************************************************/

void nvbit_at_init() {
	setenv("CUDA_MANAGED_FORCE_DEVICE_ALLOC", "1", 1);
	GET_VAR_INT(instr_begin_interval, "INSTR_BEGIN", 0,
			"Beginning of the instruction interval where to apply instrumentation");
	GET_VAR_INT(instr_end_interval, "INSTR_END", UINT32_MAX,
			"End of the instruction interval where to apply instrumentation");
	GET_VAR_INT(verbose, "TOOL_VERBOSE", 0, "Enable verbosity inside the tool");
	//std::string pad(100, '-');
	//printf("%s\n", pad.c_str());
	nvbit_trace_file.open(output_trace_file, std::ios::out);
}
/* Set used to avoid re-instrumenting the same functions multiple times */
std::unordered_set<hipFunction_t> already_instrumented;

void instrument_function_if_needed(hipCtx_t ctx, hipFunction_t func) {
	/* Get related functions of the kernel (device function that can be
	 * called by the kernel) */
	std::vector<hipFunction_t> related_functions = nvbit_get_related_functions(ctx, func);

	/* add kernel itself to the related function vector */
	related_functions.push_back(func);

	/* iterate on function */
	for (auto f : related_functions) {
		/* "recording" function was instrumented, if set insertion failed
		 * we have already encountered this function */
		if (!already_instrumented.insert(f).second) {
			continue;
		}
		const std::vector<Instr *> &instrs = nvbit_get_instrs(ctx, f);
		if (verbose) {
			printf("Inspecting function %s at address 0x%lx\n", nvbit_get_func_name(ctx, f),
					nvbit_get_func_addr(f));
		}

		uint32_t cnt = 0;
		/* iterate on all the static instructions in the function */
		for (auto instr : instrs) {
			if (cnt < instr_begin_interval || cnt >= instr_end_interval) {
				cnt++;
				continue;
			}
			if (verbose) {
				instr->printDecoded();
			}

			if (sass_to_id_map.find(instr->getSass()) == sass_to_id_map.end()) {
				int opcode_id = sass_to_id_map.size();
				sass_to_id_map[instr->getSass()] = opcode_id;
				id_to_sass_map[opcode_id] = std::string(instr->getSass());
			}

			int opcode_id = sass_to_id_map[instr->getSass()];
			std::vector<int> reg_num_list;
			/* iterate on the operands */
			for (int i = 0; i < instr->getNumOperands(); i++) {
				/* get the operand "i" */
				const InstrType::operand_t *op = instr->getOperand(i);
				if (op->type == InstrType::OperandType::REG) {
					reg_num_list.push_back(op->u.reg.num);
				}
			}
			/* insert call to the instrumentation function with its
			 * arguments */
			nvbit_insert_call(instr, "record_reg_val", IPOINT_BEFORE);
			/* guard predicate value */
			nvbit_add_call_arg_guard_pred_val(instr);
			/* opcode id */
			nvbit_add_call_arg_const_val32(instr, opcode_id);
			/* add pointer to channel_dev*/
			nvbit_add_call_arg_const_val64(instr, (uint64_t) & channel_dev);
			/* how many register values are passed next */
			nvbit_add_call_arg_const_val32(instr, reg_num_list.size());
			/**************************************************************************
			 * Edit: trying to load all the cbank values
			 **************************************************************************/
			// extract vector of pair with c[bankid][bankoffset]
			auto cbank_values = extract_cbank_vector(instr->getSass());
			// how many constant operands
			nvbit_add_call_arg_const_val32(instr, cbank_values.size());
			// For some reason I have to put the size of the operands at
			// the end of the var list
			nvbit_add_call_arg_const_val32(instr, reg_num_list.size() + cbank_values.size());

			//REGs FIRST as I will read them before the cbank values
			for (int num : reg_num_list) {
				/* last parameter tells it is a variadic parameter passed to
				 * the instrument function record_reg_val() */
				nvbit_add_call_arg_reg_val(instr, num, true);
			}

			//instrument the constant operands
			for (auto& cbank : cbank_values) {
//				std::cout << "SASS: " << instr->getSass() << " - c[" << cbank.first << "]["
//						<< cbank.second << "]\n";
				nvbit_add_call_arg_cbank_val(instr, cbank.first, cbank.second, true);
			}
			/**************************************************************************/
			cnt++;
		}
	}
}

__global__ void flush_channel() {
	/* push memory access with negative cta id to communicate the kernel is
	 * completed */
	reg_info_t ri;
	ri.cta_id_x = -1;
	channel_dev.push(&ri, sizeof(reg_info_t));

	/* flush channel */
	channel_dev.flush();
}

void nvbit_at_cuda_event(hipCtx_t ctx, int is_exit, nvbit_api_cuda_t cbid, const char *name,
		void *params, hipError_t *pStatus) {
	if (skip_flag)
		return;

	if (cbid == API_CUDA_cuLaunchKernel_ptsz || cbid == API_CUDA_cuLaunchKernel) {
		cuLaunchKernel_params *p = (cuLaunchKernel_params *) params;

		if (!is_exit) {
			int nregs;
			CUDA_SAFECALL(hipFuncGetAttribute(&nregs, HIP_FUNC_ATTRIBUTE_NUM_REGS, p->f));

			int shmem_static_nbytes;
			CUDA_SAFECALL(
					hipFuncGetAttribute(&shmem_static_nbytes, HIP_FUNC_ATTRIBUTE_SHARED_SIZE_BYTES,
							p->f));

			instrument_function_if_needed(ctx, p->f);

			nvbit_enable_instrumented(ctx, p->f, true);

//			printf("Kernel %s - grid size %d,%d,%d - block size %d,%d,%d - nregs "
//					"%d - shmem %d - cuda stream id %ld\n", nvbit_get_func_name(ctx, p->f),
//					p->gridDimX, p->gridDimY, p->gridDimZ, p->blockDimX, p->blockDimY, p->blockDimZ,
//					nregs, shmem_static_nbytes + p->sharedMemBytes, (uint64_t) p->hStream);
			recv_thread_receiving = true;

			/**
			 * Fernando mod
			 */
			nvbit_trace_file << "Kernel " << nvbit_get_func_name(ctx, p->f) << " gridsize "
					<< p->gridDimX << "," << p->gridDimY << "," << p->gridDimZ << " blocksize "
					<< p->blockDimX << "," << p->blockDimY << "," << p->blockDimZ << " nregs "
					<< nregs << " shmem " << shmem_static_nbytes + p->sharedMemBytes
					<< " cudastreamid " << (uint64_t) p->hStream << std::endl;

//			nvbit_trace_file << "cta_id_x,cta_id_y,cta_id_z,warp_id,global_warp_id,sm_id,lane_id,opcode\n";

		} else {
			/* make sure current kernel is completed */
			hipDeviceSynchronize();
			hipError_t kernelError = hipGetLastError();
			if (kernelError != hipSuccess) {
				printf("Kernel launch error: %s\n", hipGetErrorString(kernelError));
				assert(0);
			}

			/* make sure we prevent re-entry on the nvbit_callback when issuing
			 * the flush_channel kernel */
			skip_flag = true;

			/* issue flush of channel so we are sure all the memory accesses
			 * have been pushed */
			flush_channel<<<1, 1>>>();
			hipDeviceSynchronize();
			assert(hipGetLastError() == hipSuccess);

			/* unset the skip flag */
			skip_flag = false;

			/* wait here until the receiving thread has not finished with the
			 * current kernel */
			while (recv_thread_receiving) {
				pthread_yield();
			}
		}
	}
}

void print_data(reg_info_t* ri) {
	printf("CTA %d,%d,%d - warp %d - %s:\n", ri->cta_id_x, ri->cta_id_y, ri->cta_id_z, ri->warp_id,
			id_to_sass_map[ri->opcode_id].c_str());
	for (int reg_idx = 0; reg_idx < ri->num_regs; reg_idx++) {
		printf("* ");
		for (int i = 0; i < WARP_SIZE; i++) {
			printf("Reg%d_T%d: 0x%08x ", reg_idx, i, ri->reg_vals[i][reg_idx]);
		}
		printf("\n");
	}
	printf("\n");
}

void print_data_csv(reg_info_t* ri) {
//	printf("CTA %d,%d,%d - NCTA %d,%d,%d - WARPID %d - GWARPID %d - SMID %d - LANEID %d - ",
//			ri->cta_id_x, ri->cta_id_y, ri->cta_id_z, // CTA
//			ri->ncta_id_x, ri->ncta_id_y, ri->ncta_id_z, // NCTA
//			ri->warp_id, ri->global_warp_id, ri->sm_id, ri->lane_id //WARP, global WARP, SM and LANE ID
//			);

	nvbit_trace_file << "CTA " << ri->cta_id_x << "," << ri->cta_id_y << "," << ri->cta_id_z
			<< // CTA
			" NCTA " << ri->ncta_id_x << "," << ri->ncta_id_y << "," << ri->ncta_id_z
			<< // NCTA
			" WARPID " << ri->warp_id << " GWARPID " << ri->global_warp_id << " SMID " << ri->sm_id
			<< " LANEID " << ri->lane_id << " " << id_to_sass_map[ri->opcode_id] << std::endl;

//	nvbit_trace_file << ri->cta_id_x << "," << ri->cta_id_y                   << "," << ri->cta_id_z
//				<< ri->warp_id  << "," << ri->global_warp_id             << "," << ri->sm_id
//				<< ri->lane_id  << "," << id_to_sass_map[ri->opcode_id]  << ",";

//	printf("%s\n", id_to_sass_map[ri->opcode_id].c_str());
//	nvbit_trace_file << id_to_sass_map[ri->opcode_id] << std::endl;
	char temp[128];
	for (int reg_idx = 0; reg_idx < ri->num_regs; reg_idx++) {
		for (int i = 0; i < WARP_SIZE; i++) {
//			printf("R%dT%d:0x%08x ", reg_idx, i, ri->reg_vals[i][reg_idx]);
			sprintf(temp, "R%dT%d:0x%08x ", reg_idx, ri->lane_id, ri->reg_vals[i][reg_idx]);
			nvbit_trace_file << temp;
		}
		nvbit_trace_file << std::endl;
	}

	/* Print to the file the constant values */
	for (int cbank_idx = 0; cbank_idx < ri->num_cbank; cbank_idx++) {
		for (int i = 0; i < WARP_SIZE; i++) {
			sprintf(temp, "C%dT%d:0x%08x ", cbank_idx, ri->lane_id, ri->cbank_vals[i][cbank_idx]);
			nvbit_trace_file << temp;
		}
		nvbit_trace_file << std::endl;
	}
}

void *recv_thread_fun(void *) {
	char *recv_buffer = (char *) malloc(CHANNEL_SIZE);

	while (recv_thread_started) {
		uint32_t num_recv_bytes = 0;
		if (recv_thread_receiving
				&& (num_recv_bytes = channel_host.recv(recv_buffer, CHANNEL_SIZE)) > 0) {
			uint32_t num_processed_bytes = 0;
			while (num_processed_bytes < num_recv_bytes) {
				reg_info_t *ri = (reg_info_t *) &recv_buffer[num_processed_bytes];

				/* when we get this cta_id_x it means the kernel has completed
				 */
				if (ri->cta_id_x == -1) {
					recv_thread_receiving = false;
					break;
				}

				print_data_csv(ri);
				num_processed_bytes += sizeof(reg_info_t);
			}
		}
	}
	free(recv_buffer);
	return NULL;
}

void nvbit_at_ctx_init(hipCtx_t ctx) {
	recv_thread_started = true;
	channel_host.init(0, CHANNEL_SIZE, &channel_dev, NULL);
	pthread_create(&recv_thread, NULL, recv_thread_fun, NULL);
}

void nvbit_at_ctx_term(hipCtx_t ctx) {
	if (recv_thread_started) {
		recv_thread_started = false;
		pthread_join(recv_thread, NULL);
	}
}
